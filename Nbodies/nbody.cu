#include "hip/hip_runtime.h"
/*
   Running without arguments is equivalent to 1000 iterations with the
   5 celestial objects declared in the golden_bodies array.

   $ nbody.exe 1000 5

   The output of this shows the energy before and after the simulation,
   and should be:

   -0.169075164
   -0.169087605
*/

#include <chrono>
#include <cmath>
#include <cstdlib>
#include <iomanip>
#include <iostream>
#define GPUTEST 1


using type = double;

const type pi{3.141592653589793};
const type solar_mass{4 * pi * pi};
const type days_per_year{365.24};

template <typename T>
struct planet {
  T x, y, z;
  T vx, vy, vz;
  T mass;
};

//velocity update for the kernals
template <typename T>
__global__ void adv_Velocity_Update( int nbodies, planet<T> *bodies)
{

	int i = threadIdx.x + blockIdx.x*blockDim.x;
	
	if (i < nbodies)
	{

		planet<T> &b1 = bodies[i];
		for (int j = i + 1; j < nbodies; j++) 
		{
			planet<T> &b2 = bodies[j];
			T dx = b1.x - b2.x;
			T dy = b1.y - b2.y;
			T dz = b1.z - b2.z;
			T inv_distance = 1.0 / sqrt(dx * dx + dy * dy + dz * dz);
			T mag = inv_distance * inv_distance * inv_distance;
			b1.vx -= dx * b2.mass * mag;
			b1.vy -= dy * b2.mass * mag;
			b1.vz -= dz * b2.mass * mag;
			b2.vx += dx * b1.mass  * mag;
			b2.vy += dy * b1.mass  * mag;
			b2.vz += dz * b1.mass  * mag;
		}
	}
}

//position update for the kernals
template <typename T>
__global__ void adv_Position_Update( int nbodies, planet<T> *bodies)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	
	if (i < nbodies)
	{
		planet<T> &b = bodies[i];
		b.x += b.vx;
		b.y += b.vy;
		b.z += b.vz;
	}
}

template <typename T>
__global__ void scale_bodies_GPU(int nbodies, planet<T> *bodies, T scale)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;

	if (i < nbodies)
	{
		bodies[i].mass *= scale*scale;
		bodies[i].vx *= scale;
		bodies[i].vy *= scale;
		bodies[i].vz *= scale;
	}
}

template <typename T>
__global__ void energy_GPU(int nbodies, planet<T> *bodies)
{
	T e = 0.0;
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int j = i + 1;

	if (i < nbodies)
	{
		planet<T> &b = bodies[i];
		e += 0.5 * b.mass * (b.vx * b.vx + b.vy * b.vy + b.vz * b.vz);
		if (j < nbodies)
		{
			planet<T> &b2 = bodies[j];
			T dx = b.x - b2.x;
			T dy = b.y - b2.y;
			T dz = b.z - b2.z;
			T distance = sqrt(dx * dx + dy * dy + dz * dz);
			e -= (b.mass * b2.mass) / distance;
		}
	}
	//return e; need to return this type
}

template <typename T>
void advance(int nbodies, planet<T> *bodies)
{
  int i, j;

  for (i = 0; i < nbodies; ++i) {
    planet<T> &b = bodies[i];
    for (j = i + 1; j < nbodies; j++) {
      planet<T> &b2 = bodies[j];
      T dx = b.x - b2.x;
      T dy = b.y - b2.y;
      T dz = b.z - b2.z;
      T inv_distance = 1.0/sqrt(dx * dx + dy * dy + dz * dz);
      T mag = inv_distance * inv_distance * inv_distance;
      b.vx  -= dx * b2.mass * mag;
      b.vy  -= dy * b2.mass * mag;
      b.vz  -= dz * b2.mass * mag;
      b2.vx += dx * b.mass  * mag;
      b2.vy += dy * b.mass  * mag;
      b2.vz += dz * b.mass  * mag;
    }
  }
  for (i = 0; i < nbodies; ++i) {
    planet<T> &b = bodies[i];
    b.x += b.vx;
    b.y += b.vy;
    b.z += b.vz;
  }
}

template <typename T>
T energy(int nbodies, planet<T> *bodies)
{
  T e = 0.0;
  for (int i = 0; i < nbodies; ++i) {
    planet<T> &b = bodies[i];
    e += 0.5 * b.mass * (b.vx * b.vx + b.vy * b.vy + b.vz * b.vz);
    for (int j = i + 1; j < nbodies; j++) {
      planet<T> &b2 = bodies[j];
      T dx = b.x - b2.x;
      T dy = b.y - b2.y;
      T dz = b.z - b2.z;
      T distance = sqrt(dx * dx + dy * dy + dz * dz);
      e -= (b.mass * b2.mass) / distance;
    }
  }
  return e;
}

template <typename T>
void offset_momentum(int nbodies, planet<T> *bodies)
{
  T px = 0.0, py = 0.0, pz = 0.0;
  for (int i = 0; i < nbodies; ++i) {
    px += bodies[i].vx * bodies[i].mass;
    py += bodies[i].vy * bodies[i].mass;
    pz += bodies[i].vz * bodies[i].mass;
  }
  bodies[0].vx = - px / solar_mass;
  bodies[0].vy = - py / solar_mass;
  bodies[0].vz = - pz / solar_mass;
}

struct planet<type> golden_bodies[5] = {
  {                               /* sun */
    0, 0, 0, 0, 0, 0, solar_mass
  },
  {                               /* jupiter */
    4.84143144246472090e+00,
    -1.16032004402742839e+00,
    -1.03622044471123109e-01,
    1.66007664274403694e-03 * days_per_year,
    7.69901118419740425e-03 * days_per_year,
    -6.90460016972063023e-05 * days_per_year,
    9.54791938424326609e-04 * solar_mass
  },
  {                               /* saturn */
    8.34336671824457987e+00,
    4.12479856412430479e+00,
    -4.03523417114321381e-01,
    -2.76742510726862411e-03 * days_per_year,
    4.99852801234917238e-03 * days_per_year,
    2.30417297573763929e-05 * days_per_year,
    2.85885980666130812e-04 * solar_mass
  },
  {                               /* uranus */
    1.28943695621391310e+01,
    -1.51111514016986312e+01,
    -2.23307578892655734e-01,
    2.96460137564761618e-03 * days_per_year,
    2.37847173959480950e-03 * days_per_year,
    -2.96589568540237556e-05 * days_per_year,
    4.36624404335156298e-05 * solar_mass
  },
  {                               /* neptune */
    1.53796971148509165e+01,
    -2.59193146099879641e+01,
    1.79258772950371181e-01,
    2.68067772490389322e-03 * days_per_year,
    1.62824170038242295e-03 * days_per_year,
    -9.51592254519715870e-05 * days_per_year,
    5.15138902046611451e-05 * solar_mass
  }
};

const type DT{1e-2};
const type RECIP_DT{1.0/DT};

/*
 * Rescale certain properties of bodies. That allows doing
 * consequential advance()'s as if dt were equal to 1.0.
 *
 * When all advances done, rescale bodies back to obtain correct energy.
 */
template <typename T>
void scale_bodies(int nbodies, planet<T> *bodies, T scale)
{
  for (int i = 0; i < nbodies; ++i) {
    bodies[i].mass *= scale*scale;
    bodies[i].vx   *= scale;
    bodies[i].vy   *= scale;
    bodies[i].vz   *= scale;
  }
}

template <typename T>
void init_random_bodies(int nbodies, planet<T> *bodies)
{
  for (int i = 0; i < nbodies; ++i) {
    bodies[i].x    =  (T)rand()/RAND_MAX;
    bodies[i].y    =  (T)rand()/RAND_MAX;
    bodies[i].z    =  (T)rand()/RAND_MAX;
    bodies[i].vx   =  (T)rand()/RAND_MAX;
    bodies[i].vy   =  (T)rand()/RAND_MAX;
    bodies[i].vz   =  (T)rand()/RAND_MAX;
    bodies[i].mass =  (T)rand()/RAND_MAX;
  }
}

template <typename T>
void kernalUpdate(int nbodies, planet<T> *bodies)
{
	planet<T> *Gbodies;

	//Copy data from CPU
	hipMalloc(&Gbodies, nbodies*sizeof(planet<T>));
	hipMemcpy(Gbodies, bodies, nbodies, hipMemcpyHostToDevice);

	//Scaling
	hipMemcpy(Gbodies, bodies, nbodies*sizeof(planet<type>), hipMemcpyHostToDevice);
	scale_bodies_GPU << <nbodies, ceil(nbodies / 2) >> >(nbodies, Gbodies, DT);
	hipMemcpy(bodies, Gbodies, nbodies*sizeof(planet<type>), hipMemcpyDeviceToHost);

	//velocity
	hipMemcpy(Gbodies, bodies, nbodies*sizeof(planet<type>), hipMemcpyHostToDevice);
	adv_Velocity_Update<<<1, nbodies>>>(nbodies, Gbodies);
	hipMemcpy(bodies, Gbodies, nbodies*sizeof(planet<type>), hipMemcpyDeviceToHost);

	//position
	hipMemcpy(Gbodies, bodies, nbodies*sizeof(planet<type>), hipMemcpyHostToDevice);
	adv_Position_Update << <nbodies, ceil(nbodies/2)>> >(nbodies, Gbodies);
	hipMemcpy(bodies, Gbodies, nbodies*sizeof(planet<type>), hipMemcpyDeviceToHost);

	//Scaling
	hipMemcpy(Gbodies, bodies, nbodies*sizeof(planet<type>), hipMemcpyHostToDevice);
	scale_bodies_GPU << <nbodies, ceil(nbodies / 2) >> >(nbodies, Gbodies, RECIP_DT);
	hipMemcpy(bodies, Gbodies, nbodies*sizeof(planet<type>), hipMemcpyDeviceToHost);

	//copy data back to CPU
	//hipMemcpy(bodies, Gbodies, nbodies, hipMemcpyDeviceToHost);

	//Free up the memory
	hipFree(Gbodies);
}

int main(int argc, char ** argv)
{
  int niters = 1000, nbodies = 5;
  if (argc > 1) { niters  = atoi(argv[1]); }
  if (argc > 2) { nbodies = atoi(argv[2]); }

  std::cout << "niters=" << niters << " nbodies=" << nbodies << '\n';

  planet<type> *bodies;
  if (argc == 1) { 
    bodies = golden_bodies; // Check accuracy with 1000 solar system iterations
  } else {
    bodies = new planet<type>[nbodies];
    init_random_bodies(nbodies, bodies);
  }

  auto t1 = std::chrono::steady_clock::now();
  offset_momentum(nbodies, bodies);
  type e1 = energy(nbodies, bodies);
 
  if (GPUTEST)
  {
	  for (auto i = 0; i < niters; ++i)
		  kernalUpdate(nbodies, bodies);
  }
	else
	{
		  scale_bodies(nbodies, bodies, DT);
		  for (int i = 1; i <= niters; ++i)  {
		    advance(nbodies, bodies);
		  }
		  scale_bodies(nbodies, bodies, RECIP_DT);
	}
	 
  type e2 = energy(nbodies, bodies);
  auto t2 = std::chrono::steady_clock::now();
  auto diff = t2 - t1;

  std::cout << std::setprecision(9);
  std::cout << e1 << '\n' << e2 << '\n';
  std::cout << std::fixed << std::setprecision(3);
  std::cout << std::chrono::duration<double>(diff).count() << " seconds.\n";

  if (argc != 1) { delete [] bodies; }
  return 0;
}
